#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>
#include <string>

#include <assert.h>

#include "include/kernels/warp_znorm.hpp"    // cpu_znorm call
#include "include/binary_IO.hpp"
#include "include/hpc_helpers.hpp"
#include "include/cbf_generator.hpp"

#define DATABASE (0)
#define STREAM   (1)

#define TIMERSTART_CUDA(label)                                                 \
        hipSetDevice(0);                                                      \
        hipEvent_t start##label, stop##label;                                 \
        float time##label;                                                     \
        hipEventCreate(&start##label);                                        \
        hipEventCreate(&stop##label);                                         \
        hipEventRecord(start##label, 0);

#define TIMERSTOP_CUDA(label)                                                  \
        hipSetDevice(0);                                                      \
        hipEventRecord(stop##label, 0);                                       \
        hipEventSynchronize(stop##label);                                     \
        hipEventElapsedTime(&time##label, start##label, stop##label);         \
        if (query_type == DATABASE)                                            \
            std::cout << "TIMING: " << time##label << " ms " << ((num_features+1)*(num_features+1)*num_entries*num_queries)/(time##label*1e6) << " GCPUS (" << #label << ")" << std::endl; \
        if (query_type == STREAM)                                              \
            std::cout << "TIMING: " << time##label << " ms " << ((num_features+1)*(num_features+1)*(num_entries-num_features+1)*num_queries)/(time##label*1e6) << " GCPUS (" << #label << ")" << std::endl;

typedef float value_t;                              // data type for values
typedef uint64_t index_t;                           // data type for indices
typedef uint8_t  label_t;                           // data type for label

// maximum number of features fitting into constant memory
constexpr index_t max_features = (1UL<<16)/sizeof(value_t);
__constant__ value_t cQuery[max_features];

#define LOCAL_ZNORM_STREAM   // <- this triggers the local znorm in ecg 1023 kernel

#include "include/ED.hpp"
#include "include/DTW.hpp"
using namespace FullDTW;

int main (int argc, char * argv[]) {

    // choose whether this is a database query (0) or stream query (>0)
    const uint8_t query_type = DATABASE; //STREAM;

    // configure working modes
    const bool subwarp = false;    // true;                 // use subwarp kernels
    const bool enable_omp __attribute__((unused)) = false;  // enable/disable openmp in check
    const bool normalize_stream = false;                    // normalize queries in stream
    const bool lower_bound_stream = false;                  // apply lower_bounds to stream
    value_t * bsf = nullptr;

    // create systemwide pointer and initialize with infinity
    if (lower_bound_stream) {
        hipMallocManaged(&bsf, sizeof(value_t));
        *bsf = INFINITY;
    } CUERR

    const bool init_db = true;                     // initialize DB with CBF
    const bool perform_memcpy = true;              // perform data transfers

    TIMERSTART(malloc)
    index_t num_entries = 20040000; // 1UL << 21;  // entries in DB or stream
    index_t num_features = 1023;                   // length of entries
    index_t num_gpus = 2;                          // number of GPUs to be used
    index_t num_streams = 8;                       // number of streams per GPU
    index_t batch_size = 1UL << 17;                // size of a batch
    index_t buffer_size = num_streams*batch_size;  // total entries on one GPU
    index_t num_queries = 1;

    // some consistency checks
    assert(query_type == DATABASE || batch_size >= num_features);

    assert(num_features <= max_features);

    assert(num_queries == 1);

    // status
    if (query_type == DATABASE) { // query CBF database
        const value_t CU = num_features*num_features*num_entries*num_queries;
        std::cout << "We are going to process "
                  << CU/1000000000000.0
                  << " Tera Cell Updates (TCU)"
                  << std::endl;
        const value_t DM = (num_entries*num_features+num_entries)
                         * sizeof(value_t);
        std::cout << "We are going to stream exactly "
                  << DM/1073741824.0
                  << " Gibi Bytes (GiB) to and from the GPU"
                  << std::endl;

    } else {              // query ECG stream
        const value_t CU = (num_entries-num_features+1)
                         *  num_features*num_features;
        std::cout << "We are going to process "
                  << CU/1000000000000.0
                  << " Tera Cell Updates (TCU)"
                  << std::endl;
        const value_t DM = (num_entries-num_features+1)
                         * 2*sizeof(value_t);
        std::cout << "We are going to stream at least "
                  << DM/1073741824.0
                  << " Gibi Bytes (GiB) to and from the GPU"
                  << std::endl;
    }

    // create the streams on each GPU
    hipStream_t streams[num_gpus][num_streams];
    for (index_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        for (index_t stream = 0; stream < num_streams; stream++) {
            hipStreamCreate(&streams[gpu][stream]);
        }
    }
    CUERR



    value_t * data_cpu  = nullptr,                 // time series on the CPU
            * dist_cpu  = nullptr,                 // distance array on the CPU
            * data_gpu[num_gpus],                  // buffers on GPUs
            * dist_gpu[num_gpus];                  // distance arrays on GPUs

    // create host storage and buffers on devices
    if (query_type == DATABASE) { // query CBF database
        hipHostMalloc(&data_cpu, sizeof(value_t)*num_entries*num_features);
        hipHostMalloc(&dist_cpu, sizeof(value_t)*num_entries*num_queries);
        for (index_t gpu = 0; gpu < num_gpus; gpu++) {
            hipSetDevice(gpu);
            hipMalloc(&data_gpu[gpu], sizeof(value_t)*buffer_size*num_features);
            hipMalloc(&dist_gpu[gpu], sizeof(value_t)*buffer_size*num_queries);
        } CUERR
    } else {                // query ECG database
        hipHostMalloc(&data_cpu, sizeof(value_t)*num_entries);
        hipHostMalloc(&dist_cpu, sizeof(value_t)*(num_entries-num_features+1));
        for (index_t gpu = 0; gpu < num_gpus; gpu++) {
            hipSetDevice(gpu);
            hipMalloc(&data_gpu[gpu], sizeof(value_t)*buffer_size);
            hipMalloc(&dist_gpu[gpu],
                       sizeof(value_t)*num_streams*(batch_size-num_features+1));
        } CUERR
    }
    TIMERSTOP(malloc)

    TIMERSTART(generate_data)

    value_t * query_cpu = nullptr;
    hipHostMalloc(&query_cpu, sizeof(value_t)*num_features);             CUERR
    if (query_type == DATABASE) { // query CBF database
        if (init_db) {
            label_t * labels_cpu = nullptr;
            hipHostMalloc(&labels_cpu, sizeof(label_t)*num_entries);     CUERR
            generate_cbf(data_cpu, labels_cpu, num_entries, num_features);
            hipHostFree(labels_cpu);                                     CUERR

            for (index_t gpu = 0; gpu < num_gpus; gpu++) {
                hipSetDevice(gpu);
                hipMemcpyToSymbol(HIP_SYMBOL(cQuery), data_cpu,
                                   sizeof(value_t)*num_features*num_queries);
            } CUERR
        }
    } else {                // query ECG database
        if (init_db) {
            if (sizeof(value_t) == 4) {
                load_binary(data_cpu, num_entries, "data/single_subject.bin");
                load_binary(query_cpu, num_features, "data/single_query7.bin");
                #ifdef LOCAL_ZNORM_STREAM
                cpu_znorm(query_cpu, num_features);
                #endif
            }
            if (sizeof(value_t) == 8) {
                load_binary(data_cpu, num_entries, "data/double_subject.bin");
                load_binary(query_cpu, num_features, "data/double_query0.bin");
                #ifdef LOCAL_ZNORM_STREAM
                cpu_znorm(query_cpu, num_features);
                #endif
            }
            for (index_t gpu = 0; gpu < num_gpus; gpu++) {
                hipSetDevice(gpu);
                hipMemcpyToSymbol(HIP_SYMBOL(cQuery), query_cpu,
                                   sizeof(value_t)*num_features);
            } CUERR
        }
    }
    TIMERSTOP(generate_data)

    TIMERSTART_CUDA(streamed_computation)
    for (index_t batch = 0; /*no a priori bound check possible*/ ; batch++) {

        // determine gpu and stream identifier from batch identifier
        const index_t gpu = batch % num_gpus;
        const index_t stream = (batch/num_gpus) % num_streams;
        hipSetDevice(gpu);

        // range_size == batch_size in DB case but shortened by num_features
        // to account for overlap in the stream case
        const index_t range_size = query_type == DATABASE ? batch_size:
                                   batch_size-num_features+1;

        // slice the corresponding range from host memory
        const index_t lower = std::min(batch*range_size, num_entries);
        const index_t upper = std::min(lower+batch_size, num_entries);
        const index_t width = upper-lower;

        // if empty batch then exit
        if (width == 0)
            break;
        // if not enough points in last batch of stream then exit
        if (query_type == STREAM && width < num_features)
            break;

        // compute host and device pointers
        const index_t multiplicator = query_type == DATABASE ? num_features : 1;
        const auto data_ptr_gpu = data_gpu[gpu]+range_size*stream*multiplicator;
        const auto data_ptr_cpu = data_cpu     +range_size*batch*multiplicator;
        const auto dist_ptr_gpu = dist_gpu[gpu]+range_size*stream*num_queries;
        const auto dist_ptr_cpu = dist_cpu     +range_size*batch*num_queries;

        // toggle between width many time series of length num_features to be
        // copied in the DB case and width many data points in the stream case
        const index_t num_entries_data = query_type == DATABASE ?
                                         width*num_features :
                                         width;
        const index_t num_entries_dist = query_type == DATABASE?
                                         width :
                                         width-num_features+1;

        // reset score values on the GPU to 0
        hipMemsetAsync(dist_ptr_gpu, 0,
                        sizeof(value_t)*num_entries_dist*num_queries,
                        streams[gpu][stream]);

        // copy the database batch to the GPU
        if (perform_memcpy)
            hipMemcpyAsync(data_ptr_gpu, data_ptr_cpu,
                            sizeof(value_t)*num_entries_data,
                            hipMemcpyHostToDevice,
                            streams[gpu][stream]);

        // here we call the distance function
        dist(data_ptr_gpu, dist_ptr_gpu,
             width, num_features, num_queries, subwarp,
	     query_type, lower_bound_stream, bsf, streams[gpu][stream]);

        // copy distances back to CPU
         if (perform_memcpy)
            hipMemcpyAsync(dist_ptr_cpu, dist_ptr_gpu,
                            sizeof(value_t)*num_entries_dist,
                            hipMemcpyDeviceToHost,
                            streams[gpu][stream]);
    } CUERR

    // synchronize all streams
    for (index_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        for (index_t stream = 0; stream < num_streams; stream++) {
            hipStreamSynchronize(streams[gpu][stream]);
        }
    } CUERR
    TIMERSTOP_CUDA(streamed_computation)

    if (lower_bound_stream)
         std::cout << "STATUS: value stored in bsf: " << *bsf << std::endl;

    TIMERSTART(free)
    // tear down all streams and GPU memory
    for (index_t gpu = 0; gpu < num_gpus; gpu++) {
        hipSetDevice(gpu);
        for (index_t stream = 0; stream < num_streams; stream++)
            hipStreamDestroy(streams[gpu][stream]);
        hipFree(data_gpu[gpu]);
        hipFree(dist_gpu[gpu]);
    } CUERR

    if (lower_bound_stream)
        hipFree(bsf);                                                    CUERR

    // release the memory
    hipHostFree(data_cpu);                                               CUERR
    hipHostFree(dist_cpu);                                               CUERR
    hipHostFree(query_cpu);                                              CUERR
    TIMERSTOP(free)
}
